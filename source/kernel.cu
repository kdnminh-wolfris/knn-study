#include "hip/hip_runtime.h"
#include "kernel.cuh"

#ifdef __DEBUG__
#include <stdio.h>
#endif

__global__ void GetDistInd(
    float *dist, int *ind, const float *inner_prod,
    const int i_size, const int j, const int j_size,
    const float *sum_of_sqr
) {
    const int point_id = blockIdx.x * MAX_THREADS / WARP_SIZE + threadIdx.x / WARP_SIZE;
    const int lane_id = threadIdx.x % WARP_SIZE;

    if (point_id >= i_size) return;

    for (int col = lane_id; col < j_size; col += WARP_SIZE) {
        const int ij = point_id * j_size + col;
        const int jj = j * BLOCK_SIZE + col;
        dist[ij] = sum_of_sqr[jj] - 2 * inner_prod[ij];
        ind[ij] = jj;
    }
}

__global__ void AssignResults(
    const int i, const int k,
    const int row_start, const int row_stride,
    float *res_distances, int *res_indices,
    const float *dist, const int *ind, const int n_pts
) {
    // coor_init(r, c, k);
    // const int rid = (i * BLOCK_SIZE + r) * k + c;
    // const int bid = r * row_stride + row_start + c;

    // if (r >= n_pts) return;
    
    // res_distances[rid] = dist[bid];
    // res_indices[rid] = ind[bid];

    const int row_id = blockIdx.x * blockDim.x / WARP_SIZE + threadIdx.x / WARP_SIZE;
    const int lane_id = threadIdx.x % WARP_SIZE;

    if (row_id >= n_pts) return;

    const int point_ptr = (i * BLOCK_SIZE + row_id) * k;
    const int row_ptr = row_id * row_stride;

    for (int col = lane_id; col < row_start + k; col += WARP_SIZE) {
        if (col < row_start) continue;
        res_distances[point_ptr + col - row_start] = dist[row_ptr + col];
        res_indices[point_ptr + col - row_start] = ind[row_ptr + col];
    }
}

__global__ void MergeToResults(
    const int i, const int k,
    float *res_distances, int *res_indices,
    const float *dist, const int *ind,
    const int row_start, const int row_stride, const int n_pts
) {
    if (blockIdx.x * MAX_THREADS + threadIdx.x >= n_pts) return;

    const int pt = (i * BLOCK_SIZE + blockIdx.x * MAX_THREADS + threadIdx.x) * k;
    res_distances += pt;
    res_indices += pt;
    const int bp = (blockIdx.x * MAX_THREADS + threadIdx.x) * row_stride; // block point
    dist += bp + row_start;
    ind += bp + row_start;

    int p1 = 0, p2 = 0, lim2 = row_stride - row_start;
    float d1 = res_distances[0];
    float d2 = dist[0];

    while (p1 + p2 < k)
        if (p2 == lim2 || d1 <= d2) {
            if ((++p1) + p2 < k)
                d1 = res_distances[p1];
        }
        else {
            if (p1 + (++p2) < k && p2 < lim2)
                d2 = dist[p2];
        }

    d1 = res_distances[--p1];
    d2 = dist[--p2];
    for (int x = k - 1; x >= 0; --x) {
        if (p2 == -1 || (p1 > -1 && d1 > d2)) {
            res_distances[x] = d1;
            res_indices[x] = res_indices[p1--];
            if (p1 > -1) d1 = res_distances[p1];
        }
        else {
            res_distances[x] = d2;
            res_indices[x] = ind[p2--];
            if (p2 > -1) d2 = dist[p2];
        }
    }
}