#include "hip/hip_runtime.h"
#include "kernel.cuh"

#ifdef __DEBUG__
#include <stdio.h>
#endif

__global__ void __CalculateSumOfSquared(
    const int n, const int d, const float *points, float *sum_of_sqr
) {
    const int point_id = blockIdx.x * blockDim.x / WARP_SIZE + threadIdx.x / WARP_SIZE;
    if (point_id >= n) return;

    const int lane_id = threadIdx.x % WARP_SIZE;

    float sum = 0;
    for (int i = lane_id; i < d; i += WARP_SIZE)
        sum += sqr(points[point_id * d + i]);
    for (int offset = 1; offset < 32; offset <<= 1)
        sum += __shfl_down_sync(FULL_MASK, sum, offset);

    if (lane_id == 0) sum_of_sqr[point_id] = sum;
}

__global__ void __ComputeActualDistances(
    const int n, const int k, const float *sum_of_sqr, float *res_distances
) {
    const int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id / k >= n) return;

    __shared__ float sum[blockDim.x];
    const int relative_point_id = id / k - blockIdx.x * blockDim.x / k;
    if (threadIdx.x == 0 || id % k == 0)
        sum[relative_point_id] = sum_of_sqr[id / k];
    __syncthreads();

    res_distances[id] += sum[relative_point_id];
}

__global__ void __GetDistInd(
    float *dist, const float *inner_prod,
    const int i_size, const int j, const int j_size,
    const float *sum_of_sqr
) {
    // const int ii = blockIdx.x * blockDim.x / WARP_SIZE + threadIdx.x / WARP_SIZE;
    // const int lane_id = threadIdx.x % WARP_SIZE;
    // __shared__ float sum[128 * WARP_SIZE];

    // int k = 0;
    // int j_ext = intceildiv(j_size, 512) * 512;
    // for (int jj = lane_id; jj < j_ext; jj += WARP_SIZE) {
    //     if (k % 128 == 0) {
    //         __syncthreads();
    //         for (int i = threadIdx.x; i < min(j_size - k * 32, 128 * 32); i += blockDim.x) {
    //             sum[i] = sum_of_sqr[j * BLOCK_SIZE + i + k * 32];
    //         }
    //         __syncthreads();
    //     }
    //     ++k;
    //     if (ii < i_size && jj < j_size)
    //         dist[ii * j_size + jj] = sum[jj % (128 * 32)] - 2 * inner_prod[ii * j_size + jj];
    // }

    // #define ROW_SIZE 32
    // const int row_id =
    //     (blockIdx.x / intceildiv(j_size, ROW_SIZE)) * blockDim.x / ROW_SIZE
    //     + threadIdx.x / ROW_SIZE;
    // const int col_id =
    //     (blockIdx.x % intceildiv(j_size, ROW_SIZE)) * ROW_SIZE
    //     + threadIdx.x % ROW_SIZE;
    // if (row_id >= i_size || col_id >= j_size) return;
    
    // __shared__ float sum[ROW_SIZE];
    // if (threadIdx.x / ROW_SIZE == 0)
    //     sum[threadIdx.x % ROW_SIZE] = sum_of_sqr[j * BLOCK_SIZE + col_id];
    // __syncthreads();

    // const int ij = row_id * j_size + col_id;
    // dist[ij] = sum[threadIdx.x % ROW_SIZE] - 2 * inner_prod[ij];

    const int ij = blockIdx.x * MAX_THREADS + threadIdx.x;
    if (ij >= i_size * j_size) return;

    const int jj = j * BLOCK_SIZE + ij % j_size;
    dist[ij] = sum_of_sqr[jj] - 2 * inner_prod[ij];
}

__global__ void __DownHeap(
    const int k, float *heap_dist, int *heap_ind,
    const int block_i, const int block_j,
    const int i_size, const int j_size,
    const float *dist
) {
    const int data_id = blockIdx.x * blockDim.x / 32 + threadIdx.x / 32;
    const int point_id = block_i * BLOCK_SIZE + data_id;
    const int lane_id = threadIdx.x % 32;
    if (data_id >= i_size) return;
    
    heap_dist += point_id * k;
    heap_ind += point_id * k;

    for (int j = 0; j < j_size; ++j) {
        if (point_id == block_j * BLOCK_SIZE + j) continue;

        const int ptr = data_id * j_size + j;
        const float cur_dist = dist[ptr];
        if (heap_dist[0] <= dist[ptr]) continue;

        int heap_par = 0;
        int heap_ptr = 1;
        while (true) {
            // printf("%d %d %d %d %d\n", block_i, block_j, data_id, lane_id, heap_ptr);
            if (heap_ptr >= k) break;

            float max_dist = heap_ptr + lane_id >= k ? -INFINITY : heap_dist[heap_ptr + lane_id];
            int max_lane = lane_id;
            for (int offset = 1; offset < 32; offset *= 2) {
                float next_dist = __shfl_down_sync(FULL_MASK, max_dist, offset);
                int next_lane = __shfl_down_sync(FULL_MASK, max_lane, offset);
                if (max_dist < next_dist) {
                    max_dist = next_dist;
                    max_lane = next_lane;
                }
            }

            max_dist = __shfl_sync(FULL_MASK, max_dist, 0);
            if (max_dist <= cur_dist) break;

            if (lane_id == 0) {
                heap_dist[heap_par] = max_dist;
                heap_ind[heap_par] = heap_ind[heap_ptr + max_lane];
                heap_par = heap_ptr + max_lane;
                heap_ptr = heap_par * 32 + 1;
            }
            heap_par = __shfl_sync(FULL_MASK, heap_par, 0);
            heap_ptr = __shfl_sync(FULL_MASK, heap_ptr, 0);
        } // end while

        if (lane_id == 0) {
            heap_dist[heap_par] = cur_dist;
            heap_ind[heap_par] = block_j * BLOCK_SIZE + j;
        }
    }
}

__global__ void AssignInfinity(float *a) {
    a[blockIdx.x * MAX_THREADS + threadIdx.x] = INFINITY;
}