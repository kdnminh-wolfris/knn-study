#include "hip/hip_runtime.h"
#include "kernel.cuh"

#ifdef __DEBUG__
#include <stdio.h>
#endif

__global__ void __CalculateSumOfSquared(
    const int n, const int d, const float* points, float* sum_of_sqr) {
    const int data_id = blockIdx.x * MAX_THREADS / WARP_SIZE + threadIdx.x / WARP_SIZE;
    const int lane_id = threadIdx.x % WARP_SIZE;

    if (data_id >= n) return;

    float s = 0;
    for (int i = lane_id; i < d; i += WARP_SIZE)
        s += sqr(points[data_id * d + i]);
    for (int offset = 1; offset < 32; offset <<= 1)
        s += __shfl_down_sync(FULL_MASK, s, offset);
    if (lane_id == 0) sum_of_sqr[data_id] = s;
}

__global__ void __ComputeActualDistances(
    float* res_distances, const float* sum_of_sqr, const int k) {
    const int i = blockIdx.x * MAX_THREADS + threadIdx.x;

    __shared__ float sqrsum[MAX_THREADS];
    const int relative_id = i / k - blockIdx.x * MAX_THREADS / k;
    if (threadIdx.x == 0 || i % k == 0)
        sqrsum[relative_id] = sum_of_sqr[i / k];
    __syncthreads();

    res_distances[i] += sqrsum[relative_id];
}

__global__ void __GetDistInd(
    float *dist, int *ind, const float *inner_prod,
    const int i_size, const int j, const int j_size,
    const float *sum_of_sqr
) {
    const int point_id = blockIdx.x * MAX_THREADS / WARP_SIZE + threadIdx.x / WARP_SIZE;
    const int lane_id = threadIdx.x % WARP_SIZE;

    if (point_id >= i_size) return;

    for (int col = lane_id; col < j_size; col += WARP_SIZE) {
        const int ij = point_id * j_size + col;
        const int jj = j * BLOCK_SIZE + col;
        dist[ij] = sum_of_sqr[jj] - 2 * inner_prod[ij];
        ind[ij] = jj;
    }
}

__global__ void __AssignResults(
    const int i, const int k,
    const int row_start, const int row_stride,
    float *res_distances, int *res_indices,
    const float *dist, const int *ind, const int n_pts
) {
    const int row_id = blockIdx.x * blockDim.x / WARP_SIZE + threadIdx.x / WARP_SIZE;
    const int lane_id = threadIdx.x % WARP_SIZE;

    if (row_id >= n_pts) return;

    const int point_ptr = (i * BLOCK_SIZE + row_id) * k;
    const int row_ptr = row_id * row_stride;

    for (int col = lane_id; col < row_start + k; col += WARP_SIZE) {
        if (col < row_start) continue;
        res_distances[point_ptr + col - row_start] = dist[row_ptr + col];
        res_indices[point_ptr + col - row_start] = ind[row_ptr + col];
    }
}

__global__ void __MergeToResults(
    const int i, const int k,
    float *res_distances, int *res_indices,
    const float *dist, const int *ind,
    const int row_start, const int row_stride, const int n_pts
) {
    if (blockIdx.x * MAX_THREADS + threadIdx.x >= n_pts) return;

    const int pt = (i * BLOCK_SIZE + blockIdx.x * MAX_THREADS + threadIdx.x) * k;
    res_distances += pt;
    res_indices += pt;
    const int bp = (blockIdx.x * MAX_THREADS + threadIdx.x) * row_stride; // block point
    dist += bp + row_start;
    ind += bp + row_start;

    int p1 = 0, p2 = 0, lim2 = row_stride - row_start;
    float d1 = res_distances[0];
    float d2 = dist[0];

    while (p1 + p2 < k)
        if (p2 == lim2 || d1 <= d2) {
            if ((++p1) + p2 < k)
                d1 = res_distances[p1];
        }
        else {
            if (p1 + (++p2) < k && p2 < lim2)
                d2 = dist[p2];
        }

    d1 = res_distances[--p1];
    d2 = dist[--p2];
    for (int x = k - 1; x >= 0; --x) {
        if (p2 == -1 || (p1 > -1 && d1 > d2)) {
            res_distances[x] = d1;
            res_indices[x] = res_indices[p1--];
            if (p1 > -1) d1 = res_distances[p1];
        }
        else {
            res_distances[x] = d2;
            res_indices[x] = ind[p2--];
            if (p2 > -1) d2 = dist[p2];
        }
    }
}