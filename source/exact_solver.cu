#include "hip/hip_runtime.h"
#include "exact_solver.cuh"
#include <thrust/sort.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>

#ifdef __DEBUG__

#include <iostream>
#include <stdio.h>
#include <algorithm>

#endif

void KnnSolver::Solve() {
    PreProcessing();
    __Solve();
    PostProcessing();
    CleanOnDevice();
}

void KnnSolver::PreProcessing() {
    hipMalloc(&d_points, n * d * sizeof(float));
    hipMemcpy(d_points, points, n * d * sizeof(float), hipMemcpyHostToDevice);

    hipMalloc(&sum_of_sqr, n * sizeof(float));
    hipMemset(sum_of_sqr, 0, n * sizeof(float));

    CalculateSumOfSquared<<<block_cnt(n * d), MAX_THREADS>>>(n, d, d_points, sum_of_sqr);

    ResultInit();
}

__global__ void CalculateSumOfSquared(
    const int n, const int d, const float* points, float* sum_of_sqr
) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    const int pt = i / d;
    const int di = i % d;
    
    if (pt >= n) return;

    const bool flag = (threadIdx.x == 0 || di == 0);

    __shared__ float sos[MAX_THREADS];
    if (flag) sos[threadIdx.x / d] = 0;
    __syncthreads();

    const int rpt = threadIdx.x / d; // relative point id on sos array

    const float val = points[i];
    atomicAdd(&sos[rpt], val * val);
    __syncthreads();

    if (flag) sum_of_sqr[pt] += sos[rpt];
}

void KnnSolver::ResultInit() {
    res_indices = new int[n * k];
    res_distances = new float[n * k];

    hipMalloc(&d_indices, n * k * sizeof(int));
    hipMalloc(&d_distances, n * k * sizeof(float));
}

void KnnSolver::PostProcessing() {
    ComputeRealDistances<<<n, k>>>(d_distances, sum_of_sqr, k);
    hipMemcpy(res_distances, d_distances, n * k * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(res_indices, d_indices, n * k * sizeof(int), hipMemcpyDeviceToHost);
}

__global__ void ComputeRealDistances(float* res_distances, const float* sum_of_sqr, const int k) {
    __shared__ float this_sum_of_sqr;
    if (threadIdx.x == 0) this_sum_of_sqr = sum_of_sqr[blockIdx.x];
    __syncthreads();
    res_distances[blockIdx.x * k + threadIdx.x] += this_sum_of_sqr;
}

struct StartOp {
    const int m;
    __device__ int operator()(int x) const {
        return m * x;
    };
};

#ifdef __DEBUG__
void printArray(float* arr, int sizex, int sizey) {
    const int size = sizex * sizey;
    float* tmp = new float[sizex * sizey];
    hipMemcpy(tmp, arr, size * sizeof(float), hipMemcpyDeviceToHost);

    cout << sizex << ' ' << sizey << endl;
    for (int i = 0; i < size; ++i)
        cout << tmp[i] << " \n"[i % sizey == sizey - 1];
    cout << endl;
    delete[] tmp;
}
#endif

void KnnSolver::__Solve() {
    __PreProcessing();

    for (int i = 0; i < intceildiv(n, BLOCK_SIZE); ++i) {
        const int i_size = min(BLOCK_SIZE, n - i * BLOCK_SIZE);
        const float *i_block = d_points + i * BLOCK_SIZE * d;

        for (int j = 0; j < intceildiv(n, BLOCK_SIZE); ++j) {
            const int j_size = min(BLOCK_SIZE, n - j * BLOCK_SIZE);
            const float *j_block = d_points + j * BLOCK_SIZE * d;
            
            // CALCulate distances of each pair of points
            __Calc(i_size, i_block, j, j_size, j_block);

            // SORT neighbours of each point by their distance
            __Sort(i_size, j_size);

            // MERGE current k nearest neighbours of each point with the neighbours
            // which are just calculated and sorted, and keep the k nearest in the
            // result arrays
            __Merge(i, i_size, j, j_size);
        }
    }
    
    __PostProcessing();
}

void KnnSolver::__PreProcessing() {
    hipblasCreate(&handle);

    hipMalloc(&inner_prod, sqr(BLOCK_SIZE) * sizeof(float));

    hipMalloc(&dist, sqr(BLOCK_SIZE) * sizeof(float));
    hipMalloc(&ind, sqr(BLOCK_SIZE) * sizeof(int));

    hipMalloc(&tdist, sqr(BLOCK_SIZE) * sizeof(float));
    hipMalloc(&tind, sqr(BLOCK_SIZE) * sizeof(int));

    db_dist = {dist, tdist};
    db_ind = {ind, tind};

    hipHostAlloc(&tmp_d, k * sizeof(float), hipHostMallocDefault);
    hipHostAlloc(&tmp_i, k * sizeof(int), hipHostMallocDefault);
    hipMalloc(&dtmp_d, k * sizeof(float));
    hipMalloc(&dtmp_i, k * sizeof(int));
}

void KnnSolver::__PostProcessing() {
    hipblasDestroy(handle);
    hipFree(inner_prod);
    hipFree(dist);
    hipFree(ind);
    hipFree(tdist);
    hipFree(tind);
    hipHostFree(tmp_d);
    hipHostFree(tmp_i);
    hipFree(dtmp_d);
    hipFree(dtmp_i);
    if (aux) hipFree(aux);
}

void KnnSolver::__Calc(
    const int i_size, const float *i_block,
    const int j, const int j_size, const float *j_block
) {
    hipblasSgemm(
        handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
        j_size, i_size, d,
        &alpha, j_block, d, i_block, d,
        &beta, inner_prod, j_size
    );

    GetDistInd<<<block_cnt(i_size * j_size), MAX_THREADS>>>
        (db_dist.Current(), db_ind.Current(), inner_prod, i_size, j, j_size, sum_of_sqr);
}

void KnnSolver::__Sort(const int i_size, const int j_size) {
    hipcub::TransformInputIterator<int, StartOp, decltype(itr)> start_itr(itr, {j_size});

    hipcub::DeviceSegmentedSort::SortPairs(
        nullptr, aux_size,
        db_dist, db_ind,
        i_size * j_size, i_size,
        start_itr, start_itr + 1
    );
    if (aux_size > pre_aux_size) {
        if (aux) hipFree(aux);
        hipMalloc(&aux, aux_size);
        pre_aux_size = aux_size;
    }

    hipcub::DeviceSegmentedSort::SortPairs(
        aux, aux_size,
        db_dist, db_ind,
        i_size * j_size, i_size,
        start_itr, start_itr + 1
    );
}

void KnnSolver::__Merge(
    const int i, const int i_size,
    const int j, const int j_size
) {
    for (int ii = 0, i_index = i * BLOCK_SIZE; ii < i_size; ++ii, ++i_index)
        if (j == 0)
            AssignResults<<<1, k>>>(
                i == j, db_dist.Current() + ii * j_size, db_ind.Current() + ii * j_size,
                (i * BLOCK_SIZE + ii) * k, d_distances, d_indices
            );
        else
            InsertToResults(
                db_dist.Current() + ii * j_size + (i == j), db_ind.Current() + ii * j_size + (i == j),
                k, i_index, d_distances, d_indices
            );
}