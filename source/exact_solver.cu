#include "hip/hip_runtime.h"
#include "exact_solver.cuh"
#include <thrust/sort.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>

#ifdef __DEBUG__

#include <iostream>
#include <stdio.h>
#include <algorithm>

void printArray(float* arr, int sizex, int sizey) {
    const int size = sizex * sizey;
    float* tmp = new float[sizex * sizey];
    hipMemcpy(tmp, arr, size * sizeof(float), hipMemcpyDeviceToHost);

    cout << sizex << ' ' << sizey << endl;
    for (int i = 0; i < size; ++i)
        cout << tmp[i] << " \n"[i % sizey == sizey - 1];
    cout << endl;
    delete[] tmp;
}

#endif

void KnnSolver::Solve() {
    PreProcessing();
    __Solve();
    PostProcessing();
    CleanOnDevice();
}

void KnnSolver::PreProcessing() {
    hipMalloc(&d_points, n * d * sizeof(float));
    hipMemcpy(d_points, points, n * d * sizeof(float), hipMemcpyHostToDevice);

    hipMalloc(&sum_of_sqr, n * sizeof(float));
    hipMemset(sum_of_sqr, 0, n * sizeof(float));

    CalculateSumOfSquared(n, d, d_points, sum_of_sqr);

    ResultInit();
}

void KnnSolver::ResultInit() {
    res_indices = new int[n * k];
    res_distances = new float[n * k];

    hipMalloc(&d_indices, n * k * sizeof(int));
    hipMalloc(&d_distances, n * k * sizeof(float));
}

void KnnSolver::PostProcessing() {
    ComputeActualDistances(n, d_distances, sum_of_sqr, k);
    hipMemcpy(res_distances, d_distances, n * k * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(res_indices, d_indices, n * k * sizeof(int), hipMemcpyDeviceToHost);
}

void KnnSolver::__Solve() {
    __PreProcessing();

    const int n_blocks = intceildiv(n, BLOCK_SIZE);
    for (int i = 0; i < n_blocks; ++i) {
        const int i_size = min(BLOCK_SIZE, n - i * BLOCK_SIZE);
        const float *i_block = d_points + i * BLOCK_SIZE * d;

        for (int j = 0; j < n_blocks; ++j) {
            const int j_size = min(BLOCK_SIZE, n - j * BLOCK_SIZE);
            const float *j_block = d_points + j * BLOCK_SIZE * d;

            __Calc(i_size, i_block, j, j_size, j_block);
            __Push_Heap(i, j, i_size, j_size);
        }
    }
    
    __PostProcessing();
}

void KnnSolver::__PreProcessing() {
    hipblasCreate(&handle);
    hipMalloc(&inner_prod, sqr(BLOCK_SIZE) * sizeof(float));
    hipMalloc(&d_dist, sqr(BLOCK_SIZE) * sizeof(float));
    AssignInfinity(n, k, d_distances);
}

void KnnSolver::__PostProcessing() {
    hipblasDestroy(handle);
    hipFree(inner_prod);
    if (aux) hipFree(aux);
    hipFree(d_dist);
}

void KnnSolver::__Calc(
    const int i_size, const float *i_block,
    const int j, const int j_size, const float *j_block
) {
    hipblasSgemm(
        handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
        j_size, i_size, d,
        &alpha, j_block, d, i_block, d,
        &beta, inner_prod, j_size
    );

    GetDistances(j, i_size, j_size, inner_prod, sum_of_sqr, d_dist);
}

void KnnSolver::__Push_Heap(const int i, const int j, const int i_size, const int j_size) {
    PushHeap(k, d_distances, d_indices, i, j, i_size, j_size, d_dist);
}