#include "hip/hip_runtime.h"
#include "exact_solver.cuh"
#include <thrust/sort.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>

#ifdef __DEBUG__

#include <iostream>
#include <stdio.h>
#include <algorithm>

void printArray(float* arr, int sizex, int sizey) {
    const int size = sizex * sizey;
    float* tmp = new float[sizex * sizey];
    hipMemcpy(tmp, arr, size * sizeof(float), hipMemcpyDeviceToHost);

    cout << sizex << ' ' << sizey << endl;
    for (int i = 0; i < size; ++i)
        cout << tmp[i] << " \n"[i % sizey == sizey - 1];
    cout << endl;
    delete[] tmp;
}

#endif

void KnnSolver::Solve() {
    PreProcessing();
    __Solve();
    PostProcessing();
    CleanOnDevice();
}

void KnnSolver::PreProcessing() {
    hipMalloc(&d_points, n * d * sizeof(float));
    hipMemcpy(d_points, points, n * d * sizeof(float), hipMemcpyHostToDevice);

    hipMalloc(&sum_of_sqr, n * sizeof(float));
    hipMemset(sum_of_sqr, 0, n * sizeof(float));

    CalculateSumOfSquared(n, d, d_points, sum_of_sqr);

    ResultInit();
}

void KnnSolver::ResultInit() {
    res_indices = new int[n * k];
    res_distances = new float[n * k];

    hipMalloc(&d_indices, n * k * sizeof(int));
    hipMalloc(&d_distances, n * k * sizeof(float));
}

void KnnSolver::PostProcessing() {
    ComputeActualDistances(n, d_heap_dist, sum_of_sqr, k);
    hipMemcpy(res_distances, d_heap_dist, n * k * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(res_indices, d_heap_ind, n * k * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_heap_dist);
    hipFree(d_heap_ind);
}

long long cu_timer;
long long get_timer;

void KnnSolver::__Solve() {
    __PreProcessing();

    const int n_blocks = intceildiv(n, BLOCK_SIZE);
    for (int i = 0; i < n_blocks; ++i) {
        const int i_size = min(BLOCK_SIZE, n - i * BLOCK_SIZE);
        const float *i_block = d_points + i * BLOCK_SIZE * d;

        cu_timer = get_timer = 0;
        long long calc_timer = 0, heap_timer = 0;

        for (int j = 0; j < n_blocks; ++j) {
            const int j_size = min(BLOCK_SIZE, n - j * BLOCK_SIZE);
            const float *j_block = d_points + j * BLOCK_SIZE * d;
            
            auto start = chrono::high_resolution_clock::now();
            __Calc(i_size, i_block, j, j_size, j_block);
            auto stop = chrono::high_resolution_clock::now();
            calc_timer += chrono::duration_cast<chrono::nanoseconds>(stop - start).count();

            start = chrono::high_resolution_clock::now();
            __Push_Heap(i, j, i_size, j_size);
            stop = chrono::high_resolution_clock::now();
            heap_timer += chrono::duration_cast<chrono::nanoseconds>(stop - start).count();
        }

        cout << heap_timer << ' ' << calc_timer << ' ' << cu_timer << ' ' << get_timer << endl;
        // cout << calc_timer << endl;
    }
    
    __PostProcessing();
}

void KnnSolver::__PreProcessing() {
    hipblasCreate(&handle);

    hipMalloc(&inner_prod, sqr(BLOCK_SIZE) * sizeof(float));

    hipMalloc(&d_dist, sqr(BLOCK_SIZE) * sizeof(float));

    hipMalloc(&d_heap_dist, n * k * sizeof(float));
    AssignInfinity<<<linearly_distribute(n * k)>>>(d_heap_dist);

    hipMalloc(&d_heap_ind, n * k * sizeof(int));
}

void KnnSolver::__PostProcessing() {
    hipblasDestroy(handle);
    hipFree(inner_prod);
    if (aux) hipFree(aux);
    hipFree(d_dist);
}

void KnnSolver::__Calc(
    const int i_size, const float *i_block,
    const int j, const int j_size, const float *j_block
) {
    auto start = chrono::high_resolution_clock::now();
    hipblasSgemm(
        handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
        j_size, i_size, d,
        &alpha, j_block, d, i_block, d,
        &beta, inner_prod, j_size
    );
    auto stop = chrono::high_resolution_clock::now();
    cu_timer = chrono::duration_cast<chrono::nanoseconds>(stop - start).count();

    start = chrono::high_resolution_clock::now();
    GetDistInd(
        d_dist, inner_prod, i_size, j, j_size, sum_of_sqr
    );
    stop = chrono::high_resolution_clock::now();
    get_timer = chrono::duration_cast<chrono::nanoseconds>(stop - start).count();
}

void KnnSolver::__Push_Heap(const int i, const int j, const int i_size, const int j_size) {
    __DownHeap<<<block_16x32_distribute(i_size)>>>(
        k, d_heap_dist, d_heap_ind, i, j, i_size, j_size, d_dist
    );
}